#include <stdio.h>
#include <chrono> //para medir tempo de execução

#include <hip/hip_runtime.h>

/**
 * @brief Kernel para soma de dois vetores na gpu (C[i] = A[i] + B[i])
 * @param A ponteiro para vetor A na gpu
 * @param B ponteiro para vetor B na gpu
 * @param C ponteiro para vetor de resultado na gpu
 * @param numElements número de elementos dos vetores, deve ser igual para todos
 */
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    //Verifica se a thread está dentro do vetor ainda
    if(i < numElements){
        C[i] = A[i] + B[i]; 
    }
}


int main () {
    auto numElementos = 1000000; //quantidade de elementos
    size_t size = numElementos * sizeof(float); //quantidade de bytes de cada vetor

    float *mem_A = (float *) malloc(size); //Vetor A na memória principal
    float *mem_B = (float *) malloc(size); //Vetor B na memória principal
    float *mem_C = (float *) malloc(size); //Vetor C na memória principal

    //Inicializar vetores com números aleatórios
    printf("Inicializando Vetores...\n");
    for(int i = 0 ; i < numElementos ; ++i){
        mem_A[i] = rand() / (float)RAND_MAX;
        mem_B[i] = rand() / (float)RAND_MAX;
    }

    hipFree(0); //"Acorda" a GPU

    auto overhead_comeco = std::chrono::high_resolution_clock::now();
    float *gpu_A = nullptr; //Vetor A na memória da GPU
    hipMalloc((void **) &gpu_A, size);
    hipMemcpy(gpu_A, mem_A, size, hipMemcpyHostToDevice); //Copia dados do vetor A na memoria principal para GPU

    float *gpu_B = nullptr; //Vetor B na memória da GPU
    hipMalloc((void **) &gpu_B, size);
    hipMemcpy(gpu_B, mem_B, size, hipMemcpyHostToDevice); //Copia dados do vetor B na memoria principal para GPU

    float *gpu_C = nullptr; //Vetor C na memória da GPU
    hipMalloc((void **) &gpu_C, size);
    auto overhead_fim = std::chrono::high_resolution_clock::now();
    printf("Tempo para copiar dados para GPU: %lluus\n",
        std::chrono::duration_cast<std::chrono::microseconds>(overhead_fim - overhead_comeco).count());

    hipDeviceProp_t props_gpu;
    hipGetDeviceProperties(&props_gpu, 0); //Lê as propriedades da GPU

    int threadsPerBlock = props_gpu.maxThreadsPerBlock; //Quantidade de threads paralelas por bloco de execução
    int blocksPerGrid = (numElementos + threadsPerBlock - 1) / threadsPerBlock; //quantidade de blocos por lote
    printf("Executando kernel CUDA com %d blocos de %d threads para %ld elementos\n", blocksPerGrid, threadsPerBlock, numElementos);

    auto comeco = std::chrono::high_resolution_clock::now(); //Timestamp de inicio da execução

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(gpu_A, gpu_B, gpu_C, numElementos); //Executa o kernel

    auto fim = std::chrono::high_resolution_clock::now(); //Timestamp de fim da execução
    auto duracao = std::chrono::duration_cast<std::chrono::microseconds>(fim-comeco); //Diferença entre os timestamps
    printf("Tempo de execução: %lluus\n", duracao.count());

    printf("Copiando resultados...\n");
    hipMemcpy(mem_C, gpu_C, size, hipMemcpyDeviceToHost); //Copia resultados da gpu para memoria principal

    //Verifica se os resultados estão corretos
    printf("Verificando Resultados...\n");
    comeco = std::chrono::high_resolution_clock::now();

    for(int i  = 0 ; i < numElementos ; ++i){
        if(mem_A[i] + mem_B[i] != mem_C[i]){
            printf("Erro verificando resultado no elemento %d\n", i);
        }
    }

    fim = std::chrono::high_resolution_clock::now();
    duracao = std::chrono::duration_cast<std::chrono::microseconds>(fim-comeco);
    printf("Tempo de verificação: %lluus\n", duracao.count());

    //Desaloca memória
    hipFree(gpu_A);
    hipFree(gpu_B);
    hipFree(gpu_C);

    free(mem_A);
    free(mem_B);
    free(mem_C);
}